#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <ctime>
#include <sys/time.h>
#include <hip/hip_runtime.h>
#include "hipsparse.h"
#include <iostream>
#include <cmath>
#include <float.h>
//#include "anonymouslib_cuda.h"
#include "spmv_kernel.h"
using namespace std;

int spMV_mgpu_baseline(int m, int n, long long nnz, double * alpha,
				 		double * csrVal, long long * csrRowPtr, int * csrColIndex, 
				 		double * x, double * beta,
				 		double * y,
				 		int ngpu){

	double curr_time = 0.0;
	double time_parse = 0.0;
	double time_comm = 0.0;
	double time_comp = 0.0;
	double time_post = 0.0;


	curr_time = get_time();

	hipStream_t * stream = new hipStream_t [ngpu];

	hipError_t * cudaStat1 = new hipError_t[ngpu];
	hipError_t * cudaStat2 = new hipError_t[ngpu];
	hipError_t * cudaStat3 = new hipError_t[ngpu];
	hipError_t * cudaStat4 = new hipError_t[ngpu];
	hipError_t * cudaStat5 = new hipError_t[ngpu];
	hipError_t * cudaStat6 = new hipError_t[ngpu];

	hipsparseStatus_t * status = new hipsparseStatus_t[ngpu];
	hipsparseHandle_t * handle = new hipsparseHandle_t[ngpu];
	hipsparseMatDescr_t * descr = new hipsparseMatDescr_t[ngpu];

	int  * start_row  = new int[ngpu];
	int  * end_row    = new int[ngpu];
		
	int * dev_m            = new int      [ngpu];
	int * dev_n            = new int      [ngpu];
	int * dev_nnz          = new int      [ngpu];
	int ** host_csrRowPtr  = new int    * [ngpu];
	int ** dev_csrRowPtr   = new int    * [ngpu];
	int ** dev_csrColIndex = new int    * [ngpu];
	double ** dev_csrVal   = new double * [ngpu];


	double ** dev_x = new double * [ngpu];
	double ** dev_y = new double * [ngpu];

	


	for (int d = 0; d < ngpu; d++){

		hipSetDevice(d);

		start_row[d] = floor((d)     * m / ngpu);
		end_row[d]   = floor((d + 1) * m / ngpu) - 1;

		dev_m[d]   = end_row[d] - start_row[d] + 1;
		dev_n[d]   = n;

		long long nnz_ll = csrRowPtr[end_row[d] + 1] - csrRowPtr[start_row[d]];
		long long matrix_data_space = nnz_ll * sizeof(double) + 
										nnz_ll * sizeof(int) + 
										(long long)(dev_m[d]+1) * sizeof(int) + 
										(long long)dev_n[d] * sizeof(double) +
										(long long)dev_m[d] * sizeof(double);
		double matrix_size_in_gb = (double)matrix_data_space / 1e9;
		if ( matrix_size_in_gb > 0.8 * get_gpu_availble_mem(ngpu)) {
			return -1;
		}

		dev_nnz[d] = (int)(csrRowPtr[end_row[d] + 1] - csrRowPtr[start_row[d]]);
		host_csrRowPtr[d] = new int[dev_m[d] + 1];
		for (int i = 0; i < dev_m[d] + 1; i++) {
			host_csrRowPtr[d][i] = (int)(csrRowPtr[start_row[d] + i] - csrRowPtr[start_row[d]]);
		}

	}


	time_parse = get_time() - curr_time;
	curr_time = get_time();

	for (int d = 0; d < ngpu; d++){
		hipSetDevice(d);

		hipStreamCreate(&(stream[d]));
		
		status[d] = hipsparseCreate(&(handle[d])); 
		if (status[d] != HIPSPARSE_STATUS_SUCCESS) 
		{ 
			printf("CUSPARSE Library initialization failed");
			return 1; 
		} 
		status[d] = hipsparseSetStream(handle[d], stream[d]);
		if (status[d] != HIPSPARSE_STATUS_SUCCESS) 
		{ 
			printf("Stream bindind failed");
			return 1;
		} 
		status[d] = hipsparseCreateMatDescr(&descr[d]);
		if (status[d] != HIPSPARSE_STATUS_SUCCESS) 
		{ 
			printf("Matrix descriptor initialization failed");
			return 1;
		} 	
		hipsparseSetMatType(descr[d],HIPSPARSE_MATRIX_TYPE_GENERAL); 
		hipsparseSetMatIndexBase(descr[d],HIPSPARSE_INDEX_BASE_ZERO); 

		cudaStat1[d] = hipMalloc((void**)&dev_csrRowPtr[d],   (dev_m[d] + 1) * sizeof(int));
		cudaStat2[d] = hipMalloc((void**)&dev_csrColIndex[d], dev_nnz[d] * sizeof(int)); 
		cudaStat3[d] = hipMalloc((void**)&dev_csrVal[d],      dev_nnz[d] * sizeof(double)); 

		cudaStat4[d] = hipMalloc((void**)&dev_x[d],           dev_n[d] * sizeof(double)); 
		cudaStat5[d] = hipMalloc((void**)&dev_y[d],           dev_m[d] * sizeof(double)); 
		

		if ((cudaStat1[d] != hipSuccess) || 
			(cudaStat2[d] != hipSuccess) || 
			(cudaStat3[d] != hipSuccess) || 
			(cudaStat4[d] != hipSuccess) || 
			(cudaStat5[d] != hipSuccess)) 
		{ 
			printf("Device malloc failed");
			return 1; 
		} 

		//cout << "Start copy to GPUs...";
		cudaStat1[d] = hipMemcpy(dev_csrRowPtr[d],   host_csrRowPtr[d],                  (size_t)((dev_m[d] + 1) * sizeof(int)), hipMemcpyHostToDevice);
		cudaStat2[d] = hipMemcpy(dev_csrColIndex[d], &csrColIndex[csrRowPtr[start_row[d]]], (size_t)(dev_nnz[d] * sizeof(int)),   hipMemcpyHostToDevice); 
		cudaStat3[d] = hipMemcpy(dev_csrVal[d],      &csrVal[csrRowPtr[start_row[d]]],      (size_t)(dev_nnz[d] * sizeof(double)), hipMemcpyHostToDevice);
		cudaStat4[d] = hipMemcpy(dev_y[d], &y[start_row[d]], (size_t)(dev_m[d]*sizeof(double)), hipMemcpyHostToDevice); 
		cudaStat5[d] = hipMemcpy(dev_x[d], x,                (size_t)(dev_n[d]*sizeof(double)), hipMemcpyHostToDevice); 
		

		if ((cudaStat1[d] != hipSuccess) ||
		 	(cudaStat2[d] != hipSuccess) ||
		  	(cudaStat3[d] != hipSuccess) ||
		   	(cudaStat4[d] != hipSuccess) ||
		    (cudaStat5[d] != hipSuccess)) 
		{ 
			printf("Memcpy from Host to Device failed"); 
			return 1; 
		} 

	}

	time_comm = get_time() - curr_time;
	curr_time = get_time();

	for (int d = 0; d < ngpu; ++d) 
	{
		hipSetDevice(d);
		status[d] = hipsparseDcsrmv(handle[d],HIPSPARSE_OPERATION_NON_TRANSPOSE, 
								   dev_m[d], dev_n[d], dev_nnz[d], 
								   alpha, descr[d], dev_csrVal[d], 
								   dev_csrRowPtr[d], dev_csrColIndex[d], 
								   dev_x[d], beta, dev_y[d]);		 	 	
	}
	for (int d = 0; d < ngpu; ++d) 
	{
		hipSetDevice(d);
		hipDeviceSynchronize();
		if (status[d] != HIPSPARSE_STATUS_SUCCESS) {
			return -1;
		}
	}


	

	time_comp = get_time() - curr_time;
	curr_time = get_time();

	for (int d = 0; d < ngpu; d++)
	{
		hipMemcpy( &y[start_row[d]], dev_y[d], (size_t)(dev_m[d]*sizeof(double)),  hipMemcpyDeviceToHost);
	}

	for (int d = 0; d < ngpu; d++) {
		hipSetDevice(d);
		hipFree(dev_csrVal[d]);
		hipFree(dev_csrRowPtr[d]);
		hipFree(dev_csrColIndex[d]);
		hipFree(dev_x[d]);
		hipFree(dev_y[d]);
	}

	
	delete[] dev_csrVal;
	delete[] dev_csrRowPtr;
	delete[] dev_csrColIndex;
	delete[] dev_x;
	delete[] dev_y;
	delete[] host_csrRowPtr;
	delete[] start_row;
	delete[] end_row;

	time_post = get_time() - curr_time;
		
	//cout << "time_parse = " << time_parse << ", time_comm = " << time_comm << ", time_comp = "<< time_comp <<", time_post = " << time_post << endl;

	return 0;

}
