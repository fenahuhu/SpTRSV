#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include "hipsparse.h"
#include <vector>
#include <iostream>
#include <cstdio>
#include <pthread.h>
#include "spmv_task.h"
#include "spmv_kernel.h"
#include <omp.h>
//#include "anonymouslib_cuda.h"

using namespace std;

void * spmv_worker(void * arg);

void generate_tasks(int m, int n, long long nnz, double * alpha,
				    double * csrVal, long long * csrRowPtr, int * csrColIndex, 
				  	double * x, double * beta,
				  	double * y,
				  	long long nb,
				  	vector<spmv_task *> * spmv_task_pool_ptr);

void assign_task(spmv_task * t, int dev_id, hipStream_t stream);

int run_task(spmv_task * t, int dev_id, hipsparseHandle_t handle, int kernel);

void finalize_task(spmv_task * t, int dev_id, hipStream_t stream);

void gather_results(vector<spmv_task *> * spmv_task_completed, double * y, double * beta, int m);

void print_task_info(spmv_task * t);

int spMV_mgpu_v2(int m, int n, long long nnz, double * alpha,
				  double * csrVal, long long * csrRowPtr, int * csrColIndex, 
				  double * x, double * beta,
				  double * y,
				  int ngpu, 
				  int kernel,
				  long long nb,
				  int q)
{

	nb = min(nb, (long long )(0.8*get_gpu_availble_mem(ngpu)*1e9/(double)(sizeof(double) + sizeof(int) + sizeof(int)))/q ); 
	if (nb <= 0 || ngpu == 0 || q == 0) {
		return -1;
	}


	double curr_time = 0.0;
	double time_parse = 0.0;
	double time_comm_comp = 0.0;
	double time_post = 0.0;

	curr_time = get_time();

	vector<spmv_task *> * spmv_task_pool = new vector<spmv_task *>();
	vector<spmv_task *> * spmv_task_completed = new vector<spmv_task *>();

	generate_tasks(m, n, nnz, alpha,
				  csrVal, csrRowPtr, csrColIndex, 
				  x, beta, y, nb,
				  spmv_task_pool);

	int num_of_tasks = (*spmv_task_pool).size();

	(*spmv_task_completed).reserve(num_of_tasks);

	time_parse = get_time() - curr_time;

	curr_time = get_time();

	//cout << "starting " << ngpu << " GPUs." << endl;
	omp_set_num_threads(ngpu);
	//cout << "omp_get_max_threads = " << omp_get_max_threads() << endl;
	//cout << "omp_get_thread_limit = " << omp_get_thread_limit() << endl;
	#pragma omp parallel default (shared)
	{
		
		int c;
		unsigned int dev_id = omp_get_thread_num();
		//cout << "thread " << dev_id <<"/" << omp_get_num_threads() << "started" << endl;
		hipSetDevice(dev_id);
		
		hipsparseStatus_t status[q];
		hipStream_t stream[q];
		hipsparseHandle_t handle[q];



		double ** dev_csrVal = new double * [q];
		int ** dev_csrRowPtr = new int    * [q];
		int ** dev_csrColIndex = new int  * [q];
		double ** dev_x = new double      * [q];
		double ** dev_y = new double      * [q];

		for (c = 0; c < q; c++) {
			hipStreamCreate(&(stream[c]));
			status[c] = hipsparseCreate(&(handle[c])); 
			if (status[c] != HIPSPARSE_STATUS_SUCCESS) 
			{ 
				printf("CUSPARSE Library initialization failed");
				//return 1; 
			} 
			status[c] = hipsparseSetStream(handle[c], stream[c]);
			if (status[c] != HIPSPARSE_STATUS_SUCCESS) 
			{ 
				printf("Stream bindind failed");
				//return 1;
			} 

			hipMalloc((void**)&(dev_csrVal[c]),      nb      * sizeof(double));
			hipMalloc((void**)&(dev_csrRowPtr[c]),   (m + 1) * sizeof(int)   );
			hipMalloc((void**)&(dev_csrColIndex[c]), nb      * sizeof(int)   );
			hipMalloc((void**)&(dev_x[c]),           n       * sizeof(double));
	    	hipMalloc((void**)&(dev_y[c]),           m       * sizeof(double));

    	}

   		c = 0; 
    	
    	//cout << "GPU " << dev_id << " entering loop" << endl;


    	int num_of_assigned_task = 0;
    	int num_of_to_be_assigned_task = num_of_tasks * (dev_id + 1) /  omp_get_num_threads() - 
    									 num_of_tasks * (dev_id) /  omp_get_num_threads();

		while (true) {

			spmv_task * curr_spmv_task;

			for (c = 0; c < q; c++) {

				//cout << "GPU " << dev_id << " try to get one task" << endl;
				#pragma omp critical
				{

					if(num_of_assigned_task < num_of_to_be_assigned_task &&
						 (*spmv_task_pool).size() > 0) {
						curr_spmv_task = (*spmv_task_pool)[(*spmv_task_pool).size() - 1];
						(*spmv_task_pool).pop_back();
						(*spmv_task_completed).push_back(curr_spmv_task);
						num_of_assigned_task++;
						//cout << "GPU " << dev_id << " got one task" << endl;
						//cout << "Number of task left: " << (*spmv_task_pool).size() << "/" << num_of_tasks << endl;
					} else {
						curr_spmv_task = NULL;
					}
				}

				if (curr_spmv_task) {

					curr_spmv_task->dev_csrVal = dev_csrVal[c];
					curr_spmv_task->dev_csrRowPtr = dev_csrRowPtr[c];
					curr_spmv_task->dev_csrColIndex = dev_csrColIndex[c];
					curr_spmv_task->dev_x = dev_x[c];
					curr_spmv_task->dev_y = dev_y[c];
					assign_task(curr_spmv_task, dev_id, stream[c]);
					run_task(curr_spmv_task, dev_id, handle[c], kernel);
					finalize_task(curr_spmv_task, dev_id, stream[c]);
				}
				if (!curr_spmv_task) {
					break;
				}

			}
			if (!curr_spmv_task) {
				break;
			}
		}

		hipDeviceSynchronize();

		for (c = 0; c < q; c++) {

			hipFree(dev_csrVal[c]);
			hipFree(dev_csrRowPtr[c]);
			hipFree(dev_csrColIndex[c]);
			hipFree(dev_x[c]);
			hipFree(dev_y[c]);
			hipsparseDestroy(handle[c]);
			hipStreamDestroy(stream[c]);
		}

		


	}

	time_comm_comp = get_time() - curr_time;

	curr_time = get_time();

	gather_results(spmv_task_completed, y, beta, m);

	for (int t = 0; t < (*spmv_task_completed).size(); t++) {
		hipHostFree((*spmv_task_completed)[t]->host_csrRowPtr);
		hipHostFree((*spmv_task_completed)[t]->local_result_y);
		hipHostFree((*spmv_task_completed)[t]->alpha);
		hipHostFree((*spmv_task_completed)[t]->beta);

	}

	time_post = get_time() - curr_time;

	//cout << "time_parse = " << time_parse << ", time_comm_comp = " << time_comm_comp << ", time_post = " << time_post << endl;
}



void generate_tasks(int m, int n, long long nnz, double * alpha,
				    double * csrVal, long long * csrRowPtr, int * csrColIndex, 
				  	double * x, double * beta,
				  	double * y,
				  	long long nb,
				  	vector<spmv_task *> * spmv_task_pool_ptr) {

	int num_of_tasks = (int)((nnz + nb - 1) / nb);
	//cout << "num_of_tasks = " << num_of_tasks << endl;

	int curr_row;
	int t;
	int d;

	spmv_task * spmv_task_pool = new spmv_task[num_of_tasks];

	// Calculate the start and end index
	for (t = 0; t < num_of_tasks; t++) {
		long long tmp1 = t * nnz;
		long long tmp2 = (t + 1) * nnz;

		double tmp3 = (double)(tmp1 / num_of_tasks);
		double tmp4 = (double)(tmp2 / num_of_tasks);

		spmv_task_pool[t].start_idx = floor((double)(tmp1 / num_of_tasks));
		spmv_task_pool[t].end_idx   = floor((double)(tmp2 / num_of_tasks)) - 1;
		spmv_task_pool[t].dev_nnz = (int)(spmv_task_pool[t].end_idx - spmv_task_pool[t].start_idx + 1);
	}

	// Calculate the start and end row
	curr_row = 0;
	for (t = 0; t < num_of_tasks; t++) {

		spmv_task_pool[t].start_row = get_row_from_index(m, csrRowPtr, spmv_task_pool[t].start_idx);
		// Mark imcomplete rows
		// True: imcomplete
		if (spmv_task_pool[t].start_idx > csrRowPtr[spmv_task_pool[t].start_row]) {
			spmv_task_pool[t].start_flag = true;
			spmv_task_pool[t].y2 = y[spmv_task_pool[t].start_row];
		} else {
			spmv_task_pool[t].start_flag = false;
		}
	}

	curr_row = 0;
	for (t = 0; t < num_of_tasks; t++) {
		spmv_task_pool[t].end_row = get_row_from_index(m, csrRowPtr, spmv_task_pool[t].end_idx);

		// Mark imcomplete rows
		// True: imcomplete
		if (spmv_task_pool[t].end_idx < csrRowPtr[spmv_task_pool[t].end_row + 1] - 1)  {
			spmv_task_pool[t].end_flag = true;
			spmv_task_pool[t].y2 = y[spmv_task_pool[t].end_row];
		} else {
			spmv_task_pool[t].end_flag = false;
		}
	}

	// Cacluclate dimensions
	for (t = 0; t < num_of_tasks; t++) {
		spmv_task_pool[t].dev_m = spmv_task_pool[t].end_row - spmv_task_pool[t].start_row + 1;
		spmv_task_pool[t].dev_n = n;
	}


	for (t = 0; t < num_of_tasks; t++) {
		hipHostMalloc((void **)&(spmv_task_pool[t].host_csrRowPtr), (spmv_task_pool[t].dev_m + 1) * sizeof(int));

		spmv_task_pool[t].host_csrRowPtr[0] = 0;
		spmv_task_pool[t].host_csrRowPtr[spmv_task_pool[t].dev_m] = spmv_task_pool[t].dev_nnz;
	
		// memcpy(&(spmv_task_pool[t].host_csrRowPtr[1]), 
		// 	   &csrRowPtr[spmv_task_pool[t].start_row + 1], 
		// 	   (spmv_task_pool[t].dev_m - 1) * sizeof(int) );

	
		for (int j = 1; j < spmv_task_pool[t].dev_m; j++) {
			spmv_task_pool[t].host_csrRowPtr[j] = (int)(csrRowPtr[spmv_task_pool[t].start_row + j] - spmv_task_pool[t].start_idx);
		}

		spmv_task_pool[t].host_csrColIndex = csrColIndex;
		spmv_task_pool[t].host_csrVal = csrVal;
		spmv_task_pool[t].host_y = y;
		spmv_task_pool[t].host_x = x;

		hipHostMalloc((void **)&(spmv_task_pool[t].local_result_y), spmv_task_pool[t].dev_m * sizeof(double));

		hipHostMalloc((void **)&(spmv_task_pool[t].alpha), 1 * sizeof(double));

		hipHostMalloc((void **)&(spmv_task_pool[t].beta), 1 * sizeof(double));

		spmv_task_pool[t].alpha[0] = *alpha;
		spmv_task_pool[t].beta[0] = *beta;
	}

	for (t = 0; t < num_of_tasks; t++) {
		hipsparseStatus_t status = hipsparseCreateMatDescr(&(spmv_task_pool[t].descr));
		if (status != HIPSPARSE_STATUS_SUCCESS) 
		{ 
			printf("Matrix descriptor initialization failed");
			//return 1;
		} 	
		hipsparseSetMatType(spmv_task_pool[t].descr,HIPSPARSE_MATRIX_TYPE_GENERAL); 
		hipsparseSetMatIndexBase(spmv_task_pool[t].descr,HIPSPARSE_INDEX_BASE_ZERO);
	}

	(*spmv_task_pool_ptr).reserve(num_of_tasks);
	for (t = 0; t < num_of_tasks; t++) {
		(*spmv_task_pool_ptr).push_back(&spmv_task_pool[t]);
	}

}

void assign_task(spmv_task * t, int dev_id, hipStream_t stream){
	t->dev_id = dev_id;
	hipSetDevice(dev_id);

    hipMemcpyAsync(t->dev_csrRowPtr,   t->host_csrRowPtr,          
    			   (size_t)((t->dev_m + 1) * sizeof(int)), hipMemcpyHostToDevice, stream);

	hipMemcpyAsync(t->dev_csrColIndex, &(t->host_csrColIndex[t->start_idx]), 
		           (size_t)(t->dev_nnz * sizeof(int)), hipMemcpyHostToDevice, stream); 

	hipMemcpyAsync(t->dev_csrVal,      &(t->host_csrVal[t->start_idx]),
		           (size_t)(t->dev_nnz * sizeof(double)), hipMemcpyHostToDevice, stream); 

	hipMemcpyAsync(t->dev_y, &(t->host_y[t->start_row]), 
		           (size_t)(t->dev_m * sizeof(double)), hipMemcpyHostToDevice, stream); 
	
	hipMemcpyAsync(t->dev_x, t->host_x,
				   (size_t)(t->dev_n * sizeof(double)),  hipMemcpyHostToDevice, stream);

}

int run_task(spmv_task * t, int dev_id, hipsparseHandle_t handle, int kernel){
	hipSetDevice(dev_id);

	hipsparseStatus_t status;
	int err;
	if(kernel == 1) {
		status = hipsparseDcsrmv(handle, HIPSPARSE_OPERATION_NON_TRANSPOSE, 
								t->dev_m, t->dev_n, t->dev_nnz, 
								t->alpha, t->descr, t->dev_csrVal, 
								t->dev_csrRowPtr, t->dev_csrColIndex, 
								t->dev_x,  t->beta, t->dev_y); 
	} else if (kernel == 2) {
		status = cusparseDcsrmv_mp(handle, HIPSPARSE_OPERATION_NON_TRANSPOSE, 
									t->dev_m, t->dev_n, t->dev_nnz, 
									t->alpha, t->descr, t->dev_csrVal, 
									t->dev_csrRowPtr, t->dev_csrColIndex, 
									t->dev_x,  t->beta, t->dev_y); 
	} 
    
    /*
    else if (kernel == 3) {
		err = csr5_kernel(t->dev_m, t->dev_n, t->dev_nnz, 
					t->alpha, t->dev_csrVal, 
					t->dev_csrRowPtr, t->dev_csrColIndex, 
					t->dev_x,  t->beta, t->dev_y); 
		
	}
    */
	if (status != HIPSPARSE_STATUS_SUCCESS || err != 0 ) {
		return -1;
	}
}

void finalize_task(spmv_task * t, int dev_id, hipStream_t stream) {
	hipSetDevice(dev_id);
	hipMemcpyAsync(t->local_result_y,   t->dev_y,          
    			   (size_t)((t->dev_m) * sizeof(double)), 
    			   hipMemcpyDeviceToHost, stream);
}

void gather_results(vector<spmv_task *> * spmv_task_completed, double * y, double * beta, int m) {
	
	int t = 0;
	bool * flag = new bool[m];
	for (int i = 0; i < m; i++) {
		flag[i] = false;
	}
	for (t = 0; t < (*spmv_task_completed).size(); t++) {
		 //cout << "Task " << t << endl;
		 //cout << "flag = " << (*spmv_task_completed)[t]->start_flag <<" " <<   (*spmv_task_completed)[t]->end_flag << endl;
		 //for (int i = 0; i < (*spmv_task_completed)[t]->dev_m; i++) {
		 //	cout << (*spmv_task_completed)[t]->local_result_y[i] << " ";
		 //}
		 //cout << endl;

		double tmp = 0.0;

		if ((*spmv_task_completed)[t]->dev_m == 1 && 
			((*spmv_task_completed)[t]->start_flag) && 
			((*spmv_task_completed)[t]->end_flag)) {
				if (!flag[(*spmv_task_completed)[t]->start_row]) {
					flag[(*spmv_task_completed)[t]->start_row] = true;
				} else {
					tmp = y[(*spmv_task_completed)[t]->start_row];
					(*spmv_task_completed)[t]->local_result_y[0] += tmp;
					(*spmv_task_completed)[t]->local_result_y[0] -= (*beta) * (*spmv_task_completed)[t]->y2;
				}
		}
		
		else {
			if ((*spmv_task_completed)[t]->start_flag) {
				if (!flag[(*spmv_task_completed)[t]->start_row]) {
					flag[(*spmv_task_completed)[t]->start_row] = true;
				} else {
					tmp = y[(*spmv_task_completed)[t]->start_row];
					(*spmv_task_completed)[t]->local_result_y[0] += tmp;
					(*spmv_task_completed)[t]->local_result_y[0] -= (*beta) * (*spmv_task_completed)[t]->y2;
				}
			}

			if ((*spmv_task_completed)[t]->end_flag) {
				if (!flag[(*spmv_task_completed)[t]->end_row]) {
					flag[(*spmv_task_completed)[t]->end_row] = true;
				} else {
					tmp = y[(*spmv_task_completed)[t]->end_row];
					(*spmv_task_completed)[t]->local_result_y[(*spmv_task_completed)[t]->dev_m - 1] += tmp;
					(*spmv_task_completed)[t]->local_result_y[(*spmv_task_completed)[t]->dev_m - 1] -= (*beta) * (*spmv_task_completed)[t]->y2;
				}
			}
		}

		memcpy(&y[(*spmv_task_completed)[t]->start_row], 
			   (*spmv_task_completed)[t]->local_result_y, 
			  ((*spmv_task_completed)[t]->dev_m * sizeof(double))); 

	}
}

void print_task_info(spmv_task * t) {
	cout << "start_idx = " << t->start_idx << endl;
	cout << "end_idx = " << t->end_idx << endl;
}
