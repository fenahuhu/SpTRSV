#include "hip/hip_runtime.h"
#include "common.h"
#include "mmio_highlevel.h"
#include "utils.h"
#include "tranpose.h"
#include "findlevel.h"

#include "sptrsv_syncfree_serialref.h"
#include "sptrsv_syncfree_cuda.h"


int main(int argc, char ** argv)
{
    // report precision of floating-point
    printf("---------------------------------------------------------------------------------------------\n");
    char  *precision;
    if (sizeof(VALUE_TYPE) == 4)
    {
        precision = (char *)"32-bit Single Precision";
    }
    else if (sizeof(VALUE_TYPE) == 8)
    {
        precision = (char *)"64-bit Double Precision";
    }
    else
    {
        printf("Wrong precision. Program exit!\n");
        return 0;
    }

    printf("PRECISION = %s\n", precision);
    printf("Benchmark REPEAT = %i\n", BENCH_REPEAT);
    printf("---------------------------------------------------------------------------------------------\n");

    int m, n, nnzA, isSymmetricA;
    int *csrRowPtrA;
    int *csrColIdxA;
    VALUE_TYPE *csrValA;

    int nnzTR;
    int *cscRowIdxTR;
    int *cscColPtrTR;
    VALUE_TYPE *cscValTR;

    int device_id = 0;
    int rhs = 0;
    int substitution = SUBSTITUTION_FORWARD;

    // "Usage: ``./sptrsv -n (#gpu) -k (#task) -mtx A.mtx'' for LX=B on device 0"
    int argi = 1;

    // load number of GPU
    char *devstr;
    if(argc > argi)
    {
        devstr = argv[argi];
        argi++;
    }

    if (strcmp(devstr, "-n") != 0) return 0;
    
    int ngpu;
    if(argc > argi)
    {
        ngpu = atoi(argv[argi]);
        argi++;
    }

    int deviceCount;
	hipGetDeviceCount(&deviceCount);

	if (deviceCount < ngpu) {
		printf("Error: Not enough number of GPUs. Only %i available\n", deviceCount);
		return -1;
	}
	if (ngpu <= 0) {
		printf("Error: Number of GPU(s) needs to be greater than 0.\n");
		return -1;
	}

 //   printf("Using %i GPU(s).\n", ngpu);

    // load number of task
    
    char *kerstr;
    if(argc > argi)
    {
        kerstr = argv[argi];
        argi++;
    }
    
    if (strcmp(kerstr, "-k") != 0) {
     printf("Usage: ``./sptrans -n (#gpu) -k (#kpg) -csr -mtx A.mtx");
     return 0;
    }
    
    int np;  
    if(argc > argi)
    {
        np = atoi(argv[argi]);
        argi++;
    }

	if (np < ngpu) {
		printf("Error: Not enough kernal for each GPU. K < N");
		return -1;
	}

   printf("number of workload per GPU is %i.\n", np);
    
     rhs = 1;
     substitution = SUBSTITUTION_FORWARD;
    //load the matrix format
    char *matstr;
    if(argc > argi)
    {
        matstr = argv[argi];
        argi++;
    }
    printf("matstr = %s\n", matstr);

    // load matrix data from file
    char  *filename;
    if(argc > argi)
    {
        filename = argv[argi];
        argi++;
    }
    printf("-------------- %s --------------\n", filename);

    srand(time(NULL));
    if (strcmp(matstr, "-mtx") == 0)
    {
        // load mtx data to the csr format
        mmio_info(&m, &n, &nnzA, &isSymmetricA, filename);
        csrRowPtrA = (int *)malloc((m+1) * sizeof(int));
        csrColIdxA = (int *)malloc(nnzA * sizeof(int));
        csrValA    = (VALUE_TYPE *)malloc(nnzA * sizeof(VALUE_TYPE));
        mmio_data(csrRowPtrA, csrColIdxA, csrValA, filename);
        printf("input matrix A: ( %i, %i ) nnz = %i\n", m, n, nnzA);

        // extract L or U with a unit diagonal of A
        int *csrRowPtr_tmp = (int *)malloc((m+1) * sizeof(int));
        int *csrColIdx_tmp = (int *)malloc((m+nnzA) * sizeof(int));
        VALUE_TYPE *csrVal_tmp    = (VALUE_TYPE *)malloc((m+nnzA) * sizeof(VALUE_TYPE));

        int nnz_pointer = 0;
        csrRowPtr_tmp[0] = 0;
        for (int i = 0; i < m; i++)
        {
            for (int j = csrRowPtrA[i]; j < csrRowPtrA[i+1]; j++)
            {   
                if (substitution == SUBSTITUTION_FORWARD)
                {
                    if (csrColIdxA[j] < i)
                    {
                        csrColIdx_tmp[nnz_pointer] = csrColIdxA[j];
                        csrVal_tmp[nnz_pointer] = rand() % 10 + 1; //csrValA[j]; 
                        nnz_pointer++;
                    }
                }
                else if (substitution == SUBSTITUTION_BACKWARD)
                {
                    if (csrColIdxA[j] > i)
                    {
                        csrColIdx_tmp[nnz_pointer] = csrColIdxA[j];
                        csrVal_tmp[nnz_pointer] = rand() % 10 + 1; //csrValA[j]; 
                        nnz_pointer++;
                    }
                }
            }

            // add dia nonzero
            csrColIdx_tmp[nnz_pointer] = i;
            csrVal_tmp[nnz_pointer] = 1.0;
            nnz_pointer++;

            csrRowPtr_tmp[i+1] = nnz_pointer;
        }

        int nnz_tmp = csrRowPtr_tmp[m];
        nnzTR = nnz_tmp;

        if (substitution == SUBSTITUTION_FORWARD)
            printf("A's unit-lower triangular L: ( %i, %i ) nnz = %i\n", m, n, nnzTR);
        else if (substitution == SUBSTITUTION_BACKWARD)
            printf("A's unit-upper triangular U: ( %i, %i ) nnz = %i\n", m, n, nnzTR);

        csrColIdx_tmp = (int *)realloc(csrColIdx_tmp, sizeof(int) * nnzTR);
        csrVal_tmp = (VALUE_TYPE *)realloc(csrVal_tmp, sizeof(VALUE_TYPE) * nnzTR);

        cscRowIdxTR = (int *)malloc(nnzTR * sizeof(int));
        cscColPtrTR = (int *)malloc((n+1) * sizeof(int));
        memset(cscColPtrTR, 0, (n+1) * sizeof(int));
        cscValTR    = (VALUE_TYPE *)malloc(nnzTR * sizeof(VALUE_TYPE));

        // transpose from csr to csc
        matrix_transposition(m, n, nnzTR,
                             csrRowPtr_tmp, csrColIdx_tmp, csrVal_tmp,
                             cscRowIdxTR, cscColPtrTR, cscValTR);

        // keep each column sort 
        for (int i = 0; i < n; i++)
        {
            quick_sort_key_val_pair<int, int>(&cscRowIdxTR[cscColPtrTR[i]],
                                              &cscRowIdxTR[cscColPtrTR[i]],
                                              cscColPtrTR[i+1]-cscColPtrTR[i]);
        }

        // check unit diagonal
        int dia_miss = 0;
        for (int i = 0; i < n; i++)
        {
            bool miss;
            if (substitution == SUBSTITUTION_FORWARD)
                miss = cscRowIdxTR[cscColPtrTR[i]] != i;
            else if (substitution == SUBSTITUTION_BACKWARD)
                cscRowIdxTR[cscColPtrTR[i+1] - 1] != i;

            if (miss) dia_miss++;
        }
        //printf("dia miss = %i\n", dia_miss);
        if (dia_miss != 0) 
        {
            printf("This matrix has incomplete diagonal, #missed dia nnz = %i\n", dia_miss); 
            return;
        }

        free(csrColIdx_tmp);
        free(csrVal_tmp);
        free(csrRowPtr_tmp);

        free(csrColIdxA);
        free(csrValA);
        free(csrRowPtrA);
    }
    else if (strcmp(matstr, "-csc") == 0)
    {
        FILE *f;
        int returnvalue;

        if ((f = fopen(filename, "r")) == NULL)
            return -1;

        returnvalue = fscanf(f, "%d", &m);
        returnvalue = fscanf(f, "%d", &n);
        returnvalue = fscanf(f, "%d", &nnzTR);

        cscColPtrTR = (int *)malloc((n+1) * sizeof(int));
        memset(cscColPtrTR, 0, (n+1) * sizeof(int));
        cscRowIdxTR = (int *)malloc(nnzTR * sizeof(int));
        cscValTR    = (VALUE_TYPE *)malloc(nnzTR * sizeof(VALUE_TYPE));

        // read row idx
        for (int i = 0; i < n+1; i++)
        {
            returnvalue = fscanf(f, "%d", &cscColPtrTR[i]);
            cscColPtrTR[i]--; // from 1-based to 0-based
        }

        // read col idx
        for (int i = 0; i < nnzTR; i++)
        {
            returnvalue = fscanf(f, "%d", &cscRowIdxTR[i]);
            cscRowIdxTR[i]--; // from 1-based to 0-based
        }

        // read val
        for (int i = 0; i < nnzTR; i++)
        {
            cscValTR[i] = rand() % 10 + 1;
            //returnvalue = fscanf(f, "%lg", &cscValTR[i]);
        }

        if (f != stdin)
            fclose(f);

        // keep each column sort 
        for (int i = 0; i < n; i++)
        {
            quick_sort_key_val_pair<int, int>(&cscRowIdxTR[cscColPtrTR[i]],
                                              &cscRowIdxTR[cscColPtrTR[i]],
                                              cscColPtrTR[i+1]-cscColPtrTR[i]);
        }

        if (substitution == SUBSTITUTION_FORWARD)
            printf("Input csc unit-lower triangular L: ( %i, %i ) nnz = %i\n", m, n, nnzTR);
        else if (substitution == SUBSTITUTION_BACKWARD)
            printf("Input csc unit-upper triangular U: ( %i, %i ) nnz = %i\n", m, n, nnzTR);
       
        // check unit diagonal
        int dia_miss = 0;
        for (int i = 0; i < n; i++)
        {
            bool miss;
            if (substitution == SUBSTITUTION_FORWARD)
                miss = cscRowIdxTR[cscColPtrTR[i]] != i;
            else if (substitution == SUBSTITUTION_BACKWARD)
                cscRowIdxTR[cscColPtrTR[i+1] - 1] != i;

            if (miss) dia_miss++;
        }
        //printf("dia miss = %i\n", dia_miss);
        if (dia_miss != 0) 
        {
            printf("This matrix has incomplete diagonal, #missed dia nnz = %i\n", dia_miss); 
            return;
        }
    }

    // find level sets
    int nlevel = 0;
    int parallelism_min = 0;
    int parallelism_avg = 0;
    int parallelism_max = 0;
    findlevel_csc(cscColPtrTR, cscRowIdxTR, cscValTR, m, n, nnzTR, &nlevel,
                  &parallelism_min, &parallelism_avg, &parallelism_max);
    double fparallelism = (double)m/(double)nlevel;
    printf("This matrix/graph has %i levels, its parallelism is %4.2f (min: %i ; avg: %i ; max: %i )\n", 
           nlevel, fparallelism, parallelism_min, parallelism_avg, parallelism_max);

    // x and b are all row-major
    VALUE_TYPE *x_ref = (VALUE_TYPE *)malloc(sizeof(VALUE_TYPE) * n * rhs);
    for ( int i = 0; i < n; i++)
        for (int j = 0; j < rhs; j++)
            x_ref[i * rhs + j] = rand() % 10 + 1; //j + 1;

    VALUE_TYPE *b = (VALUE_TYPE *)malloc(sizeof(VALUE_TYPE) * m * rhs);
    VALUE_TYPE *x = (VALUE_TYPE *)malloc(sizeof(VALUE_TYPE) * n * rhs);

    for (int i = 0; i < m * rhs; i++)
        b[i] = 0;

    for (int i = 0; i < n * rhs; i++)
        x[i] = 0;

    // run csc spmv to generate b
    for (int i = 0; i < n; i++)
    {
        for (int j = cscColPtrTR[i]; j < cscColPtrTR[i+1]; j++)
        {
            int rowid = cscRowIdxTR[j]; //printf("rowid = %i\n", rowid);
            for (int k = 0; k < rhs; k++)
            {
                b[rowid * rhs + k] += cscValTR[j] * x_ref[i * rhs + k];
            }
        }
    }

   //  // set device
   // hipSetDevice(device_id);
   // hipDeviceProp_t deviceProp;
   // hipGetDeviceProperties(&deviceProp, device_id);
   //printf("---------------------------------------------------------------------------------------------\n");
   //printf("Device [ %i ] %s @ %4.2f MHz\n", device_id, deviceProp.name, deviceProp.clockRate * 1e-3f);

    // run serial syncfree SpTRSV as a reference
    printf("---------------------------------------------------------------------------------------------\n");
    sptrsv_syncfree_serialref(cscColPtrTR, cscRowIdxTR, cscValTR, m, n, nnzTR,
                              substitution, rhs, x, b, x_ref);


    // run cuda syncfree SpTRSV or SpTRSM

    printf("---------------------------------------------------------------------------------------------\n");
    
    
   double gflops_autotuned = 0;
    
    sptrsv_syncfree_cuda(cscColPtrTR, cscRowIdxTR, cscValTR, m, n, nnzTR,
                       substitution, rhs, OPT_WARP_AUTO, x, b, x_ref, &gflops_autotuned, ngpu, np);

    printf("---------------------------------------------------------------------------------------------\n");

    // done!
    free(cscRowIdxTR);
    free(cscColPtrTR);
    free(cscValTR);

    free(x);
    free(x_ref);
    free(b);

    return 0;
}

